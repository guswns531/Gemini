#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>


__global__
void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}
struct Param { int n; float a; float *x; float *y;};

int main(void)
{
	//hipDeviceptr_t dptr;
	//hipMalloc(&dptr, 1024);
	 hipSetDevice(0);
  int N = 1<<20;
  float *x, *y, *d_x, *d_y;
  x = (float*)malloc(1024*N*sizeof(float));
  y = (float*)malloc(1024*N*sizeof(float));

  hipMalloc(&d_x, 1024*N*sizeof(float));
  hipMalloc(&d_y, 1024*N*sizeof(float));
  size_t pitch_a, pitch_b, pitch_c;
  float *a;
  hipMallocPitch((void**) &a, &pitch_a, sizeof(float) * 10, 10);
  float *b;
  hipMallocManaged(&b, 1024*sizeof(float));

  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);
  // Perform SAXPY on 1M elements
  saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);
  saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);
  saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);
  saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);
  saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);
  saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);
  
  Param param;
  param.n = N;
  param.a = 2.0f;
  param.x = d_x;
  param.y = d_y;
  void *kArgs = {&param};
  dim3 gridDim;
  gridDim.x = (N+255)/256;
  
  dim3 blockDim;
  blockDim.x = 256;
  //hipLaunchCooperativeKernel((void *)saxpy, gridDim, blockDim, (void**)(N, 2.0f, d_x, d_y), 0, 0);
  
  hipDeviceSynchronize();
  hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);

  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = max(maxError, abs(y[i]-4.0f));
  printf("Max error: %f\n", maxError);

  hipFree(d_x);
  hipFree(d_y);
  free(x);
  free(y);
  hipArray* arr;

      //Create Channel Descriptor. float is just for example. Change it to required data type.
      hipChannelFormatDesc channel = hipCreateChannelDesc<float>();

      //Allocate Memory
      hipMallocArray(&arr,&channel,5, 1,hipArrayDefault);
  hipChannelFormatDesc desc = hipCreateChannelDesc<float>();

	hipExtent extent;

	extent.width = 1; // Note, for cudaArrays the width field is the width in elements, not bytes

	extent.height = 2;

	extent.depth = 3;

	hipArray *array = 0;

	hipMalloc3DArray(&array,&desc,extent,hipArrayLayered);
  hipFreeArray(arr);
  hipFreeArray(array);

  hipMipmappedArray_t* mip = 0;
  hipChannelFormatDesc des = hipCreateChannelDesc<float>();
  hipMallocMipmappedArray(mip, &des, extent, 2, 0);
  hipFreeMipmappedArray(*mip);

  
}
